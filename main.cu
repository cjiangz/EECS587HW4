
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>      /* printf, fgets */
#include <cstdlib>     /* atoi */
#include <cmath>
#include <iomanip>

using namespace std;

__constant__ int n;

__device__ int getGlobalIdx_2D_1D(){
    int blockId = blockIdx.y + (blockIdx.x + 1) * gridDim.y;
    int threadId = blockId * blockDim.x + threadIdx.x;
    return threadId;
}

__device__ int getGlobalId(int i, int j){
    return i*gridDim.y*blockDim.x + j;
}

__device__ bool isBoundary(){
    return (blockIdx.y == 0 && threadIdx.x == 0)
           || (blockIdx.y * blockDim.x + threadIdx.x >= n - 1);
}

__global__ void DoIter(double* A_old, double* A_new)
{
    __shared__ double block[1024];
    int myGlobalId = getGlobalIdx_2D_1D();
    int myLocalId = threadIdx.x;
    bool is_boundary = isBoundary();
    double temp = A_old[myGlobalId];
    block[myLocalId] = temp;
    __syncthreads();

    double above = A_old[myGlobalId - blockDim.x * gridDim.y];
    double below = A_old[myGlobalId + blockDim.x * gridDim.y];
    double left = myLocalId > 0? block[myLocalId - 1] : A_old[myGlobalId - 1];
    double right = myLocalId < 1023 ? block[myLocalId + 1] : A_old[myGlobalId + 1];
    double array[5] = {above,below,left,right,temp};

    for(int i=0;i < 3;++i) {
        int min_idx = i;
        for (int j = i; j < 5; ++j) {
            min_idx = array[min_idx] > array[j] ? j : min_idx;
        }
        double tmp = array[i];
        array[i] = array[min_idx];
        array[min_idx] = tmp;
    }

    if(!is_boundary) A_new[myGlobalId] = array[2];
}

__global__ void getVerificationValues(double* A, double* check1, double* check2){
    int myGlobalId = getGlobalIdx_2D_1D();
    int n_over_3_id = getGlobalId(n/3,n/3);
    int nineteen_thirtyseven_id = getGlobalId(19,37);
    if(myGlobalId == n_over_3_id){
        *check1 = A[myGlobalId];
    }
    if(myGlobalId == nineteen_thirtyseven_id){
        *check2 = A[myGlobalId];
    }
}

__global__ void reduce(double* A_old, double* A_new, int N) {
    __shared__ double sdata[1024];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = i >= N ? 0 : A_old[i];
    __syncthreads();

    for (unsigned int s=512; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) A_new[blockIdx.x] = sdata[0];
}

int main(int argc, char** argv)
{
    int n_ = atoi(argv[1]);
    hipMemcpyToSymbol(HIP_SYMBOL(n),&n_,sizeof(n_));
    int rounded_n = n_;
    if(n_ % 1024 != 0){
        rounded_n = 1024*(n_/1024 + 1);
    }

    double *A = new double[n_* rounded_n];
    double *A_new = new double[n_ *rounded_n];
    double sumCheck;
    double check1;
    double check2;
    for(int i = 0; i < n_; ++i){
        for(int j=0;j< rounded_n;++j){
            if(j >= n_){
                A[i*rounded_n + j] = 0;
                continue;
            }
            A[i*rounded_n + j] = sin(i*i+j)*sin(i*i+j)+cos(i-j);
        }
    }

    dim3 gridDim(n_ - 2,rounded_n/1024,1);
    dim3 blockDim(1024,1,1);

    double* d_Aold;
    double* d_Anew;
    double *d_check1;
    double *d_check2;

    if(hipMalloc(&d_Aold,sizeof(double)*n_*rounded_n) != hipSuccess){
        cout<<"Could not allocate d_A"<<endl;
    }
    if(hipMalloc(&d_Anew,sizeof(double)*n_*rounded_n) != hipSuccess){
        cout<<"Could not allocate d_A"<<endl;
    }
    if(hipMalloc(&d_check1,sizeof(double)) != hipSuccess){
        cout<<"Could not allocate d_A"<<endl;
    }
    if(hipMalloc(&d_check2,sizeof(double)) != hipSuccess){
        cout<<"Could not allocate d_A"<<endl;
    }

    if(hipMemcpy(d_Aold,A,sizeof(double)*n_*rounded_n,hipMemcpyHostToDevice) != hipSuccess){
        cout<<"Could not copy A into d_Aold"<<endl;
    }
    if(hipMemcpy(d_Anew,A,sizeof(double)*n_*rounded_n,hipMemcpyHostToDevice) != hipSuccess){
        cout<<"Could not copy A into d_Anew"<<endl;
    }

    float elapsedTime = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    for(int i = 0; i < 10; ++i){
        swap(d_Aold, d_Anew);
        DoIter<<<gridDim,blockDim>>>(d_Aold,d_Anew);
    }
    dim3 gridDim2(rounded_n,rounded_n/1024,1);
    dim3 blockDim2(1024,1,1);
    getVerificationValues<<<gridDim2,blockDim2>>>(d_Anew, d_check1, d_check2);

    int N = n_*rounded_n;
    while (N > 1) {
        swap(d_Aold, d_Anew);
        int rounded_N = N;
        if (N % 1024 != 0) {
            rounded_N = 1024 * (N / 1024 + 1);
        }
        reduce<<<rounded_N / 1024, 1024>>>(d_Aold, d_Anew, N);
        N = rounded_N/1024;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    if(hipMemcpy(A_new,d_Anew,sizeof(double)*n_*rounded_n,hipMemcpyDeviceToHost) != hipSuccess){
        cout<<"Could not copy d_Anew into A_new"<<endl;
        hipError_t error = hipGetLastError();
        cout << hipGetErrorString(error) << endl;
        cout << hipGetErrorName(error) << endl;
    }

    if(hipMemcpy(&check1, d_check1, sizeof(double), hipMemcpyDeviceToHost) != hipSuccess){
        cout<<"Could not copy d_check1 into check1"<<endl;
    }
    if(hipMemcpy(&check2, d_check2, sizeof(double), hipMemcpyDeviceToHost) != hipSuccess){
        cout<<"Could not copy d_check2 into check2"<<endl;
    }
    if(hipMemcpy(&sumCheck, d_Anew, sizeof(double), hipMemcpyDeviceToHost) != hipSuccess){
        cout<<"Could not copy d_Anew[0] into sumCheck"<<endl;
    }
    cout << std::setprecision(10) << "Sum: " << sumCheck <<  ",  A[n/3][n/3]:" << check1  << ", A[19][37]:" << check2 << ", Elapsed Time: " << elapsedTime << endl;

    delete[] A;
    delete[] A_new;
    hipFree(d_Anew);
    hipFree(d_Aold);
    hipFree(d_check1);
    hipFree(d_check2);

    return 0;
}